/*
 * buffer_impl.h
 *
 *  Created on: Dec 28, 2015
 *      Author: agibsonccc
 */


#include <buffer.h>

namespace nd4j {
namespace buffer {
/**
 *
 * @param buffer
 * @return
 */
template <typename T>
__device__ __host__ size_t bufferSize(Buffer<T> *buffer) {
	return sizeof(T) * buffer->length;
}


/**
 *
 * @param buffer
 */
template <typename T>
__host__ void copyDataToGpu(Buffer<T> **buffer) {
	Buffer<T> *bufferRef = *buffer;
	checkCudaErrors(hipMemcpy(bufferRef->gData,bufferRef->data,bufferSize(bufferRef),hipMemcpyHostToDevice));
}

/**
 *
 * @param buffer
 */
template <typename T>
__host__ void copyDataFromGpu(Buffer<T> **buffer) {
	Buffer<T> *bufferRef = *buffer;
	int bufferTotalSize = bufferSize(bufferRef);
	checkCudaErrors(hipMemcpy(bufferRef->data,bufferRef->gData,bufferTotalSize,hipMemcpyDeviceToHost));
}

/**
 * Allocate buffer of the given
 * length on the cpu and gpu.
 */
template <typename T>
__host__ void allocBuffer(Buffer<T> **buffer,int length) {
	Buffer<T> *bufferRef = *buffer;
	bufferRef->length = length;
	bufferRef->data = (T *) malloc(sizeof(T) * length);
	checkCudaErrors(hipMalloc(&bufferRef->gData,sizeof(T) * length));

}



/**
 * Frees the given buffer
 * (gpu and cpu
 */
template <typename T>
__host__ void freeBuffer(Buffer<T> **buffer) {
	Buffer<T> *bufferRef = *buffer;
	delete[] bufferRef->data;
	checkCudaErrors(hipFree(bufferRef->gData));
}




/**
 * Creates a buffer
 * based on the data
 * and also synchronizes
 * the data on the gpu.
 */
template <typename T>
__host__ Buffer<T> *createBuffer(T *data,int length) {
	Buffer<T> *ret = (Buffer<T> *) malloc(sizeof(Buffer<T>));
	ret->data = data;
	ret->length = length;
	T *gData;
	T** gDataRef = &(gData);
	checkCudaErrors(hipMalloc((void** ) gDataRef,sizeof(T) * length));
	ret->gData = gData;
	checkCudaErrors(hipMemcpy(ret->gData,ret->data,sizeof(T) * length,hipMemcpyHostToDevice));
	return ret;
}


template <typename T>
__host__ void printArr(Buffer<T> *buff) {
	for(int i = 0; i < buff->length; i++) {
		printf("Buffer[%d] was %f\n",i,buff->data[i]);
	}
}

}
}


