#include "hip/hip_runtime.h"
#include <transform.h>

template<> __device__ double op<double>(double d1,double *params) {
	return log(d1);
}


template<> __device__ float op<float>(float d1,float *params) {
	return logf(d1);
}

