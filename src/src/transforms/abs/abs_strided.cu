#include "hip/hip_runtime.h"
#include <transform.h>


template<> __device__ float op<float>(float d1,float *params) {
	return fabsf(d1);
}


template<> __device__ double op<double>(double d1,double *params) {
	return abs(d1);
}

