#include "hip/hip_runtime.h"
#include <transform.h>


template<> __device__ double op<double>(double d1,double *params) {
        return cos(d1);
}




template<> __device__ float op<float>(float d1,float *params) {
        return cosf(d1);
}


