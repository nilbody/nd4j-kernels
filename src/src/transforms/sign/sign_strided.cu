#include <transform.h>


template<> __device__ double op<double>(double d1,double *params) {
	return (d1 > 0) - (d1 < 0);
}




template<> __device__ float op<float>(float d1,float *params) {
	return (d1 > 0) - (d1 < 0);
}

