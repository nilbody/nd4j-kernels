#include "hip/hip_runtime.h"
/*
 * semaphore_impl.h
 *
 *  Created on: Dec 28, 2015
 *      Author: agibsonccc
 */

#include <semaphore.h>


inline __device__  CudaSpinLock::CudaSpinLock(int *p) {
	m_p = p;
}

inline __device__ void CudaSpinLock::acquire() {
	while (atomicCAS(m_p,0,1));
}

inline __device__ void
CudaSpinLock::release() {
	atomicExch(m_p, 0);
}




